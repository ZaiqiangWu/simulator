#include "hip/hip_runtime.h"

#include "cuda_simulation.h"
#include "spring.h"
#include "load_obj.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <iostream>
#include <fstream>
using namespace std;

extern GLenum GL_MODE;
extern bool SAVE_OBJ;

__global__ void get_face_normal(glm::vec4* g_pos_in, unsigned int* cloth_index, const unsigned int cloth_index_size, glm::vec3* cloth_face);   //update cloth face normal
__global__ void verlet(glm::vec4* pos_vbo, glm::vec4 * g_pos_in, glm::vec4 * g_pos_old_in, glm::vec4 * g_pos_out, glm::vec4 * g_pos_old_out,glm::vec4* const_pos,
						s_spring* neigh1, s_spring* neigh2,
					  glm::vec3* p_normal, unsigned int* vertex_adjface, glm::vec3* face_normal,
					  const unsigned int NUM_VERTICES,
					  BRTreeNode*  leaf_nodes, BRTreeNode*  internal_nodes, Primitive* primitives,glm::vec3* collision_force, int* collided_vertex,
					glm::vec3* d_force,glm::vec3* d_velocity, float timestep);  //verlet intergration

CUDA_Simulation::CUDA_Simulation()
{
	
}

CUDA_Simulation::~CUDA_Simulation()
{
	
}

CUDA_Simulation::CUDA_Simulation(Obj& cloth, Springs& springs):readID(0), writeID(1),sim_cloth(&cloth),NUM_ADJFACE(20),cuda_spring(&springs),dt(1/20.0)
{
	hipError_t cudaStatus = hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, sim_cloth->vbo.array_buffer, cudaGraphicsMapFlagsWriteDiscard);   	//register vbo
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "register failed\n");

	get_vertex_adjface();     //����λ��init_cudaǰ�棬������������Ϊ��
	init_cuda();              //��������ݴ���GPU


		
}

void CUDA_Simulation::simulate()
{
	size_t num_bytes;
	hipError_t cudaStatus = hipGraphicsMapResources(1, &cuda_vbo_resource, 0);
	cudaStatus = hipGraphicsResourceGetMappedPointer((void **)&cuda_p_vertex, &num_bytes, cuda_vbo_resource);
	cuda_p_normal = (glm::vec3*)((float*)cuda_p_vertex + 4 * sim_cloth->uni_vertices.size() + 2 * sim_cloth->uni_tex.size());   // ��ȡnormalλ��ָ��

	//cuda kernel compute .........
	verlet_cuda();
	cudaStatus = hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
	swap_buffer();
}

void CUDA_Simulation::init_cuda()
{
	size_t heap_size = 256 * 1024 * 1024;  //set heap size, the default is 8M
	hipDeviceSetLimit(hipLimitMallocHeapSize, heap_size);

	//��sim_cloth�ĵ�����귢�͵�GPU
	hipError_t cudaStatus;      
	const unsigned int vertices_bytes = sizeof(glm::vec4) * sim_cloth->uni_vertices.size();
	cudaStatus = hipMalloc((void**)&const_cuda_pos, vertices_bytes); // cloth vertices (const)
	cudaStatus = hipMalloc((void**)&X[0], vertices_bytes);			 // cloth vertices
	cudaStatus = hipMalloc((void**)&X[1], vertices_bytes);			 // cloth vertices
	cudaStatus = hipMalloc((void**)&X_last[0], vertices_bytes);	 // cloth old vertices
	cudaStatus = hipMalloc((void**)&X_last[1], vertices_bytes);	 // cloth old vertices
	cudaStatus = hipMalloc((void**)&collision_force, sizeof(glm::vec3) * sim_cloth->uni_vertices.size());  //collision response force
	hipMemset(collision_force, 0, sizeof(glm::vec3) * sim_cloth->uni_vertices.size());    //initilize to 0

	X_in = X[readID];
	X_out = X[writeID];
	X_last_in = X_last[readID];
	X_last_out = X_last[writeID];

	cudaStatus = hipMemcpy(const_cuda_pos, &sim_cloth->uni_vertices[0], vertices_bytes, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(X[0], &sim_cloth->uni_vertices[0], vertices_bytes, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(X_last[0], &sim_cloth->uni_vertices[0], vertices_bytes, hipMemcpyHostToDevice);

	//����normal��������ݣ�ÿ�����ڽӵ�������� + ÿ�����3��������� + �Լ����е����������ȻOPENGL�и����ݣ�
	const unsigned int vertices_index_bytes = sizeof(unsigned int) * sim_cloth->vertex_index.size();       //�������
	cudaStatus = hipMalloc((void**)&cuda_vertex_index, vertices_index_bytes);	
	cudaStatus = hipMemcpy(cuda_vertex_index, &sim_cloth->vertex_index[0], vertices_index_bytes, hipMemcpyHostToDevice);

	const unsigned int face_normal_bytes = sizeof(glm::vec3) * sim_cloth->faces.size();    //��ķ�����
	cudaStatus = hipMalloc((void**)&cuda_face_normal, face_normal_bytes);

	const unsigned int vertex_adjface_bytes = sizeof(unsigned int) * vertex_adjface.size();  //ÿ�����ڽӵ��������
	cudaStatus = hipMalloc((void**)&cuda_vertex_adjface, vertex_adjface_bytes);
	cudaStatus = hipMemcpy(cuda_vertex_adjface, &vertex_adjface[0], vertex_adjface_bytes, hipMemcpyHostToDevice);
	
	//������Ϣ���������������Ϣ����GPU
	cuda_neigh1 = cuda_spring->cuda_neigh1;
	cuda_neigh2 = cuda_spring->cuda_neigh2;

#ifdef _DEBUG
	hipMalloc((void**)&collided_vertex, sizeof(int)*sim_cloth->uni_vertices.size());
	hipMemset(collided_vertex, 0, sizeof(int)*sim_cloth->uni_vertices.size());
	cpu_collided_veretx.resize(sim_cloth->uni_vertices.size());
	updated_vertex.resize(sim_cloth->uni_vertices.size());
#endif
	updated_vertex.resize(sim_cloth->uni_vertices.size());
	cudaStatus = hipMalloc((void**)&d_force, sizeof(glm::vec3)*sim_cloth->uni_vertices.size());
	cudaStatus = hipMalloc((void**)&d_velocity, sizeof(glm::vec3)*sim_cloth->uni_vertices.size());

}

void CUDA_Simulation::get_vertex_adjface()
{
	vector<vector<unsigned int>> adjaceny(sim_cloth->uni_vertices.size());
	for(int i=0;i<sim_cloth->faces.size();i++)
	{
		unsigned int f[3];
		for(int j=0;j<3;j++)
		{
			f[j] = sim_cloth->faces[i].vertex_index[j];
			adjaceny[f[j]].push_back(i);
		}
	}

	//test
	/*for(int i=0;i<10;i++)
	{
		for(int j=0;j<adjaceny[i].size();j++)
			cout << adjaceny[i][j] << "  ";
		cout << endl;
		
	}
*/
	vertex_adjface.resize(sim_cloth->uni_vertices.size()*NUM_ADJFACE);
	for(int i=0;i<adjaceny.size();i++)
	{
		int j;
		for(j=0;j<adjaceny[i].size() && j<NUM_ADJFACE;j++)
		{
			vertex_adjface[i*NUM_ADJFACE+j] = adjaceny[i][j];
		}
		if(NUM_ADJFACE>adjaceny[i].size())
			vertex_adjface[i*NUM_ADJFACE+j] = UINT_MAX;                  //Sentinel
	}
}

void CUDA_Simulation::verlet_cuda()
{
	hipError_t cudaStatus;
	unsigned int numThreads0, numBlocks0;
	computeGridSize(sim_cloth->faces.size(), 512, numBlocks0, numThreads0);
	unsigned int cloth_index_size = sim_cloth->vertex_index.size(); 
	get_face_normal <<<numBlocks0, numThreads0 >>>(X_in, cuda_vertex_index, cloth_index_size, cuda_face_normal);  
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "normal hipDeviceSynchronize returned error code %d after launching addKernel!\n%s\n", cudaStatus, hipGetErrorString(cudaStatus));

	
	unsigned int numThreads, numBlocks;
	unsigned int numParticles = sim_cloth->uni_vertices.size();
	

	computeGridSize(numParticles, 512, numBlocks, numThreads);
	verlet <<< numBlocks, numThreads >>>(cuda_p_vertex, X_in, X_last_in, X_out, X_last_out,const_cuda_pos,
										cuda_neigh1,cuda_neigh2,
										cuda_p_normal,cuda_vertex_adjface,cuda_face_normal,
										numParticles,
										d_leaf_nodes,d_internal_nodes,d_primitives, collision_force,
										collided_vertex,
										d_force,d_velocity,dt);

	// stop the CPU until the kernel has been executed
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "verlet hipDeviceSynchronize returned error code %d after launching addKernel!\n%s\n",
			cudaStatus, hipGetErrorString(cudaStatus));
		exit(-1);
	}

	if (SAVE_OBJ)
	{
		SAVE_OBJ = false;
		hipMemcpy(&updated_vertex[0], cuda_p_vertex, sizeof(glm::vec4)*numParticles, hipMemcpyDeviceToHost);
		ofstream outfile("../tem/cloth.obj");

		outfile << "# vertices" << endl;
		for (auto ver : updated_vertex)
		{
			outfile << "v " << ver.x << " " << ver.y << " " << ver.z << endl;   //����д���ļ�
		}
		
		outfile << "# faces" << endl;
		for (auto face : sim_cloth->faces)
		{
			outfile << "f " << face.vertex_index[0]+1 << " " << face.vertex_index[1]+1 << " " << face.vertex_index[2]+1 << endl;
		}

		outfile.close();
	}
}

void CUDA_Simulation::computeGridSize(unsigned int n, unsigned int blockSize, unsigned int &numBlocks, unsigned int &numThreads)
{
	numThreads = min(blockSize, n);
	numBlocks = (n % numThreads != 0) ? (n / numThreads + 1) : (n / numThreads);
}

void CUDA_Simulation::swap_buffer()
{
	int tmp = readID;
	readID = writeID;
	writeID = tmp;

	X_in = X[readID];
	X_out = X[writeID];
	X_last_in = X_last[readID];
	X_last_out = X_last[writeID];

}

void CUDA_Simulation::add_bvh(BVHAccel& bvh)
{
	d_leaf_nodes = bvh.d_leaf_nodes;
	d_internal_nodes = bvh.d_internal_nodes;
	d_primitives = bvh.d_primitives;
}

void CUDA_Simulation::draw_collided_vertex()
{

	//draw outline first
		for (int i = 0; i < sim_cloth->faces.size(); i++)
		{
			glm::vec4 ver[3];
			glm::vec3 normal[3];
			for (int j = 0; j < 3; j++)
			{
				ver[j] = updated_vertex[sim_cloth->faces[i].vertex_index[j]];
			}
			glPointSize(1.0);
			glBegin(GL_MODE);
			glColor3f(1.0, 1.0,1.0);
			for (int j = 0; j < 3; j++)
			{
				glVertex3f(ver[j].x, ver[j].y, ver[j].z);
			}
				
			glEnd();
		}


	for (int i = 0; i < cpu_collided_veretx.size(); i++)
	{
		glm::vec4 v = updated_vertex[i];
		if (cpu_collided_veretx[i] == 1)
		{
			//draw it
			glPointSize(10.0);
			glBegin(GL_POINTS);
				glColor3f(1.0, 0, 0);
				glVertex3f(v.x, v.y, v.z);
			glEnd();
		}
	}
}